#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 100000000

__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%100000000;
	}
}

int main(void) 
{
	struct timeval t1, t2;
    long elapsed;
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}

	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	// start timer
	gettimeofday(&t1, NULL); 
	// Launch add() kernel on GPU with N blocks
	// Alloc space for host copies of a, b, c and 
	add<<<N,1>>>(d_a, d_b, d_c);
	// stop timer
	gettimeofday(&t2, NULL);
	

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	elapsed = (t2.tv_sec * 1000000 + t2.tv_usec)-(t1.tv_sec * 1000000 + t1.tv_usec);
	printf("The sum of two arrays took: %ld milliseconds\n", elapsed);

	return 0;
}